#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THRESHOLD 25  // Threshold for switching to Insertion Sort
#define MAX_STACK_SIZE 4096  // Maximum stack size for iterative quicksort

// Insertion Sort (Device Function)
__device__ void insertionSort(int *arr, int left, int right) {
    for (int i = left + 1; i <= right; i++) {
        int key = arr[i];
        int j = i - 1;
        while (j >= left && arr[j] > key) {
            arr[j + 1] = arr[j];
            j--;
        }
        arr[j + 1] = key;
    }
}

// Partition Function for Quick Sort (Device Function)
__device__ int partition(int *arr, int low, int high) {
    int pivot = arr[high];
    int i = (low - 1);
    for (int j = low; j < high; j++) {
        if (arr[j] < pivot) {
            i++;
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
    int temp = arr[i + 1];
    arr[i + 1] = arr[high];
    arr[high] = temp;
    return (i + 1);
}

// Iterative Quick Sort (Device Function)
__device__ void iterativeQuickSort(int *arr, int low, int high) {
    if (low >= high) return; // Avoid unnecessary sorting

    int stack[MAX_STACK_SIZE];
    int top = -1;

    stack[++top] = low;
    stack[++top] = high;

    while (top >= 0) {
        high = stack[top--];
        low = stack[top--];

        // If the size of the current partition is small, use insertion sort
        if (high - low < THRESHOLD) {
            insertionSort(arr, low, high);
            continue;  // Skip to the next iteration
        }

        int pi = partition(arr, low, high);

        // Push left side to stack
        if (pi - 1 > low) {
            stack[++top] = low;
            stack[++top] = pi - 1;
        }
        // Push right side to stack
        if (pi + 1 < high) {
            stack[++top] = pi + 1;
            stack[++top] = high;
        }
    }
}

// Kernel for Parallel Quick Sort
__global__ void parallelQuickSort(int *arr, int size, int segmentSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread works on a segment of the array
    int start = idx * segmentSize;
    int end = min(start + segmentSize - 1, size - 1);

    // Ensure valid indices before sorting
    if (start <= end) {
        iterativeQuickSort(arr, start, end);
    }
}

// Kernel function to perform K-Way Merge using Min-Heap
__global__ void kWayMerge(int *arr, int *temp, int size, int segmentSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread merges its own segments based on K-Way merging
    int start = idx * segmentSize * 2;
    int end1 = start + segmentSize - 1;
    int start2 = end1 + 1;
    int end2 = start2 + segmentSize - 1;

    if (start < size) {
        if (end1 >= size) end1 = size - 1;
        
        if (start2 < size) {
            if (end2 >= size) end2 = size - 1;

            int i = start, j = start2, k = start;
            while (i <= end1 && j <= end2) {
                temp[k++] = (arr[i] <= arr[j]) ? arr[i++] : arr[j++];
            }
            while (i <= end1) {
                temp[k++] = arr[i++];
            }
            while (j <= end2) {
                temp[k++] = arr[j++];
            }
        }
    }
}

void writeArrayToFile(int *arr, int size) {
    FILE *file = fopen("sorted.txt", "w");
    if (file == NULL) {
        printf("Error opening file!\n");
        return;
    }

    for (int i = 0; i < size; i++) {
        fprintf(file, "%d\n", arr[i]);
    }

    fclose(file);
    printf("Sorted output written to sorted.txt\n");
}

int main() {
    int SIZE;
    printf("Enter the number of elements: ");
    scanf("%d", &SIZE);

    int *h_arr = (int *)malloc(SIZE * sizeof(int));
    int *d_arr, *d_temp;

    // Initialize the array with random values
    srand(time(0));  // Seed for random number generation
    for (int i = 0; i < SIZE; i++) {
        h_arr[i] = rand() % 5000000;
    }

    // Print the unsorted array
    printf("Unsorted array:\n");
    for (int i = 0; i < SIZE; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // Allocate device memory with error checking
    hipError_t err;
    
    err = hipMalloc((void **)&d_arr, SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory for d_arr: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_temp, SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory for d_temp: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy host array to device
    hipMemcpy(d_arr, h_arr, SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

   // Kernel configuration
   int threadsPerBlock = 256;
   int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;

   // Determine segment size for each thread to work on
   int segmentSize = (SIZE + blocksPerGrid * threadsPerBlock - 1) / (blocksPerGrid * threadsPerBlock);

   hipEventRecord(start, 0);
   // Parallel Quick Sort
   parallelQuickSort<<<blocksPerGrid, threadsPerBlock>>>(d_arr, SIZE, segmentSize);
   hipDeviceSynchronize();  // Ensure sorting is complete

   // Stop timing for Quick Sort
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);

   // Calculate the elapsed time for Quick Sort only
   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   float seconds = milliseconds / 1000;

   // Merging sorted segments with doubling segment size using K-Way Merge
   int currentSegmentSize = segmentSize;
   while (currentSegmentSize < SIZE) {
       int mergeBlocks = (SIZE + 2 * currentSegmentSize - 1) / (2 * currentSegmentSize);
       kWayMerge<<<mergeBlocks, threadsPerBlock>>>(d_arr, d_temp, SIZE, currentSegmentSize);
       hipMemcpy(d_arr, d_temp, SIZE * sizeof(int), hipMemcpyDeviceToDevice);
       hipDeviceSynchronize(); // Ensure all threads have finished before copying back
       currentSegmentSize *= 2;  // Double the segment size for the next merging step
   }

   // Copy sorted array back to host
   hipMemcpy(h_arr, d_arr, SIZE * sizeof(int), hipMemcpyDeviceToHost);

   // Print sorted array
   printf("Sorted array:\n");
   for (int i = 0; i < SIZE; i++) {
       printf("%d ", h_arr[i]);
   }
   printf("\n");

   printf("Total Kernel execution time: %f seconds\n", seconds);

   writeArrayToFile(h_arr, SIZE);

   // Free memory
   free(h_arr);
   hipFree(d_arr);
   hipFree(d_temp);
   hipEventDestroy(start);
   hipEventDestroy(stop);

   return 0;
}
